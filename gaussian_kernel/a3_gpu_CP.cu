#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

void printArray(const float* x, int n)
{
    printf("(");
    for (int i = 0; i < n; i++)
    {
        printf("%f, ", x[i]);
    }
    printf(")\n");
}


__global__
void f_h(const int n, const float h, const float *x, float *y, bool *run)
{
    *run = true;
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float coef = 1 / (n * h) * .3989422804;
    for (int j = 0; j < n; j++)
    {
        float sum = 0;
        float x_val = x[j];
        for (int i = 0; i < n; i++)
        {
            float val = (x_val-x[i]) / h;
            float k = exp(-(val * val) / 2);
            sum = sum + k;
        }
        y[j] = coef * sum;
    }
}

__host__
void gpuCall(int n, float h, const float *x_v, float *y_v)
{
    printf("START GPU CALL\n");
    
    float *x, *y;
    bool *run;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&run, sizeof(bool));

    *run = false;

    for (int i = 0; i < n; i++)
    {
        x[i] = x_v[i];
    }

    //==============================================================
    printf("X before\n");
    printArray(x, n);
    printf("\n");

    printf("Y before\n");
    printArray(y, n);
    //==============================================================

    f_h<<<1, 1>>>(n, h, x, y, run);
    hipDeviceSynchronize();

    printf("Did it run? %d\n", *run);

    //==============================================================
    printf("\n");
    printf("Y\n");
    printArray(y, n);
    //==============================================================

    hipFree((float*)x);
    hipFree(y);
    hipFree(run);
}
