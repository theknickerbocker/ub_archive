#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

void printArray(const float* x, int n)
{
    std::cout << "(";
    for (int i = 0; i < n; i++)
    {
        std::cout << x[i] << ", ";
    }
    std::cout << ")" << std::endl;
}

// My attempt at using shared mem among blocks. Runs slightly slower than my naïve
// algorithm did but I like this more as it is at least an attempt at optimization
// even though it runs much slower than it should had it.
__global__
void f_h(const int n, const float h, const float *x, float *y, int memSize)
{
    extern __shared__ float x_reg[];
   
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const float coef = 1 / (n * h) * .3989422804;
    float sum = 0;
    float x_val = x[idx];

    for (int i = 0; i < n; i += memSize)
    {
        for (int j = 0; j < memSize; j += blockDim.x)
        {
            if(i + j + threadIdx.x < n)
            {
                x_reg[j + threadIdx.x] = x[j + i + threadIdx.x];
            }
        }
        __syncthreads();

        if (idx >= n)
        {
            return;
        }
        
        for (int k = 0; k < memSize && k+i < n; k++)
        {
            float val = (x_val-x_reg[k]) / h;
            float k_x = exp(-(val * val) / 2);
            sum = sum + k_x;
        }
    }
    y[idx] = coef * sum;
}

void gpuCall(int n, float h, const float *x_v, float *y_v)
{
    int arrSize = n*sizeof(float);

    float *x, *y;
    hipMalloc(&x, arrSize);
    hipMalloc(&y, arrSize);

    hipMemcpy(x, x_v, arrSize, hipMemcpyHostToDevice);
    hipMemcpy(y, y_v, arrSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    int memSize = blockSize * 4;

    f_h<<<numBlocks, blockSize, memSize * sizeof(float)>>>(n, h, x, y, memSize);
    hipDeviceSynchronize();

    hipMemcpy(y_v, y, arrSize, hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);
}
